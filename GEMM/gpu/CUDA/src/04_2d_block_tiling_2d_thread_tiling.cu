#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.cuh"
#include "cuda_gemm_utils.hpp"

// GEMM kernel v04.
// Coalesced read and write from global memory.
template <typename T, size_t BLOCK_TILE_SIZE_M, size_t BLOCK_TILE_SIZE_N,
          size_t BLOCK_TILE_SIZE_K, size_t THREAD_TILE_SIZE_M, size_t THREAD_TILE_SIZE_N, size_t NUM_THREADS>
__global__ void gemm_v04(size_t m, size_t n, size_t k, T alpha, T const* A,
                         size_t lda, T const* B, size_t ldb, T beta, T* C, size_t ldc) {
    
    __shared__ T A_block_tile[BLOCK_TILE_SIZE_M][BLOCK_TILE_SIZE_K];
    __shared__ T B_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_N];

    // Compute the A's and B's block tile index, which is same among all threads within a block
    // size_t const M_block_tile_id{blockIdx.y};
    // size_t const N_block_tile_id{blockIdx.x};
    // size_t const K_block_tile_num{(k + BLOCK_TILE_SIZE_K - 1) / BLOCK_TILE_SIZE_K};

    // size_t const M_thread_tile_id{threadIdx.y};
    // size_t const N_thread_tile_id{threadIdx.x};

    size_t const threadId{threadIdx.y * blockDim.x + threadIdx.x};
    size_t constexpr N_thread_tile_num{BLOCK_TILE_SIZE_N / THREAD_TILE_SIZE_N};

    size_t const M_thread_tile_index_start{threadId / N_thread_tile_num * THREAD_TILE_SIZE_M};
    size_t const N_thread_tile_index_start{threadId % N_thread_tile_num * THREAD_TILE_SIZE_N};

    T C_thread_tile[THREAD_TILE_SIZE_M][THREAD_TILE_SIZE_N] = {static_cast<T>(0)};
    T A_thread_tile[THREAD_TILE_SIZE_M] = {static_cast<T>(0)};
    T B_thread_tile[THREAD_TILE_SIZE_N] = {static_cast<T>(0)};
    //Move K_block tile in the matrix A and matrix B

    for (size_t K_block_tile_id{0}; K_block_tile_id < (k + BLOCK_TILE_SIZE_K - 1) / BLOCK_TILE_SIZE_K; K_block_tile_id++) {
        // Load A and B into block_tile,
        // and be careful to handle BLOCK_TILE_SIZE_M != BLOCK_TILE_SIZE_N
        //      and BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_K != BLOCK_TILE_SIZE_N * BLOCK_TILE_SIZE_K
        load_data_from_global_memory_to_shared_memory<T, BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K, NUM_THREADS>(
            A, lda, B, ldb, A_block_tile, B_block_tile, K_block_tile_id, threadId, m, n, k);
            
        __syncthreads();
        #pragma unroll
        for (size_t k_block_tile_idx{0}; k_block_tile_idx < BLOCK_TILE_SIZE_K; ++k_block_tile_idx) {
            // load data from shared memory to register
            // Load A_block_tile into A_thread_tile
            #pragma unroll
            for (size_t m_thread_tile_idx{0}; m_thread_tile_idx < THREAD_TILE_SIZE_M; ++m_thread_tile_idx) {
                A_thread_tile[m_thread_tile_idx] = A_block_tile[M_thread_tile_index_start + m_thread_tile_idx][k_block_tile_idx];
            }
            // Load B_block_tile into B_thread_tile
            #pragma unroll
            for (size_t n_thread_tile_idx{0}; n_thread_tile_idx < THREAD_TILE_SIZE_N; ++n_thread_tile_idx) {
                B_thread_tile[n_thread_tile_idx] = B_block_tile[k_block_tile_idx][N_thread_tile_index_start + n_thread_tile_idx];
            }

            // Compute the outer product
            for (size_t m_thread_tile_idx{0}; m_thread_tile_idx < THREAD_TILE_SIZE_M; ++m_thread_tile_idx) {
                for (size_t n_thread_tile_idx{0}; n_thread_tile_idx < THREAD_TILE_SIZE_N; ++n_thread_tile_idx) {
                    // Compute the sum
                    C_thread_tile[m_thread_tile_idx][n_thread_tile_idx] += A_thread_tile[m_thread_tile_idx] * B_thread_tile[n_thread_tile_idx];
                }
            }
            
        }
        __syncthreads();
    }
    size_t const A_block_tile_id{blockIdx.y};
    size_t const B_block_tile_id{blockIdx.x};

    // Store the result
    #pragma unroll
    for (size_t m_thread_tile_idx{0}; m_thread_tile_idx < THREAD_TILE_SIZE_M; ++m_thread_tile_idx) {
        #pragma unroll
        for (size_t n_thread_tile_idx{0}; n_thread_tile_idx < THREAD_TILE_SIZE_N; ++n_thread_tile_idx) {
            size_t C_idx_M{m_thread_tile_idx + M_thread_tile_index_start + A_block_tile_id * BLOCK_TILE_SIZE_M};
            size_t C_idx_N{n_thread_tile_idx + N_thread_tile_index_start + B_block_tile_id * BLOCK_TILE_SIZE_N};
            if (C_idx_M < m && C_idx_N < n)
                C[C_idx_M * ldc + C_idx_N] = alpha * C_thread_tile[m_thread_tile_idx][n_thread_tile_idx] + beta * C[C_idx_M * ldc + C_idx_N];
        }
    } 
}

template <typename T>
void launch_gemm_kernel_v04(size_t m, size_t n, size_t k, T const* alpha,
                            T const* A, size_t lda, T const* B, size_t ldb,
                            T const* beta, T* C, size_t ldc,
                            hipStream_t stream) {
    // Feel free to play with the block tile sizes.
    // The algorithm correctness should always be guaranteed.
    constexpr unsigned int BLOCK_TILE_SIZE_M{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_N{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_K{16U};
    
    constexpr unsigned int THREAD_TILE_SIZE_M{8U};
    constexpr unsigned int THREAD_TILE_SIZE_N{8U};                     

    constexpr unsigned int NUM_THREADS{BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_N / (THREAD_TILE_SIZE_N * THREAD_TILE_SIZE_M)};

    static_assert(NUM_THREADS <= 1024U);
    static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_N % NUM_THREADS == 0U);
    static_assert(BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_K % NUM_THREADS == 0U);
    static_assert(BLOCK_TILE_SIZE_N % THREAD_TILE_SIZE_N == 0U);
    static_assert(BLOCK_TILE_SIZE_M % THREAD_TILE_SIZE_M == 0U);
    static_assert(NUM_THREADS % THREAD_TILE_SIZE_M == 0U);
    static_assert(NUM_THREADS % THREAD_TILE_SIZE_N == 0U);

    dim3 const block_dim{NUM_THREADS, 1U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + BLOCK_TILE_SIZE_N - 1U) / BLOCK_TILE_SIZE_N,
        (static_cast<unsigned int>(m) + BLOCK_TILE_SIZE_M - 1U) / BLOCK_TILE_SIZE_M, 1U};
    gemm_v04<T, BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K, THREAD_TILE_SIZE_M, THREAD_TILE_SIZE_N, NUM_THREADS>
        <<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B, ldb, *beta, C, ldc);
    CHECK_LAST_CUDA_ERROR();
}

// Explicit instantiation.
template void launch_gemm_kernel_v04<float>(size_t m, size_t n, size_t k,
                                            float const* alpha, float const* A,
                                            size_t lda, float const* B,
                                            size_t ldb, float const* beta,
                                            float* C, size_t ldc,
                                            hipStream_t stream);
template void launch_gemm_kernel_v04<double>(size_t m, size_t n, size_t k,
                                             double const* alpha,
                                             double const* A, size_t lda,
                                             double const* B, size_t ldb,
                                             double const* beta, double* C,
                                             size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v04<__half>(size_t m, size_t n, size_t k,
                                             __half const* alpha,
                                             __half const* A, size_t lda,
                                             __half const* B, size_t ldb,
                                             __half const* beta, __half* C,
                                             size_t ldc, hipStream_t stream);