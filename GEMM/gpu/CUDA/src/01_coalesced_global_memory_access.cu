#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.hpp"

// GEMM kernel v01.
// Coalesced read and write from global memory.
template <typename T>
__global__ void gemm_v01(size_t m, size_t n, size_t k, T alpha, T const* A,
                         size_t lda, T const* B, size_t ldb, T beta, T* C,
                         size_t ldc)
{
    // Compute the row and column of C that this thread is responsible for.
    size_t const C_col_idx{blockIdx.x * blockDim.x + threadIdx.x};
    size_t const C_row_idx{blockIdx.y * blockDim.y + threadIdx.y};

    // Each thread compute
    // C[C_row_idx, C_col_idx] = alpha * A[C_row_idx, :] * B[:, C_col_idx] +
    // beta * C[C_row_idx, C_col_idx].
    if (C_row_idx < m && C_col_idx < n)
    {
        T sum{static_cast<T>(0)};
        for (size_t k_idx{0U}; k_idx < k; ++k_idx)
        {
            sum += A[C_row_idx * lda + k_idx] * B[k_idx * ldb + C_col_idx];
        }
        C[C_row_idx * ldc + C_col_idx] =
            alpha * sum + beta * C[C_row_idx * ldc + C_col_idx];
    }
}

template <typename T>
void launch_gemm_kernel_v01(size_t m, size_t n, size_t k, T const* alpha,
                            T const* A, size_t lda, T const* B, size_t ldb,
                            T const* beta, T* C, size_t ldc,
                            hipStream_t stream)
{
    dim3 const block_dim{32U, 32U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + block_dim.x - 1U) / block_dim.x,
        (static_cast<unsigned int>(m) + block_dim.y - 1U) / block_dim.y, 1U};
    gemm_v01<T><<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B,
                                                     ldb, *beta, C, ldc);
    CHECK_LAST_CUDA_ERROR();
}

// Explicit instantiation.
template void launch_gemm_kernel_v01<float>(size_t m, size_t n, size_t k,
                                            float const* alpha, float const* A,
                                            size_t lda, float const* B,
                                            size_t ldb, float const* beta,
                                            float* C, size_t ldc,
                                            hipStream_t stream);
template void launch_gemm_kernel_v01<double>(size_t m, size_t n, size_t k,
                                             double const* alpha,
                                             double const* A, size_t lda,
                                             double const* B, size_t ldb,
                                             double const* beta, double* C,
                                             size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v01<__half>(size_t m, size_t n, size_t k,
                                             __half const* alpha,
                                             __half const* A, size_t lda,
                                             __half const* B, size_t ldb,
                                             __half const* beta, __half* C,
                                             size_t ldc, hipStream_t stream);