#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.cuh"
#include "cuda_gemm_utils.hpp"

// GEMM kernel v05.
// Coalesced read and write from global memory.
template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
          size_t BLOCK_TILE_SIZE_K, size_t THREAD_TILE_SIZE_X,
          size_t THREAD_TILE_SIZE_Y>
__global__ void gemm_v05_vectorized(size_t m, size_t n, size_t k, T alpha,
                                    T const* A, size_t lda, T const* B,
                                    size_t ldb, T beta, T* C, size_t ldc)
{
    // Avoid using blockDim.x * blockDim.y as the number of threads per block.
    // Because it is a runtime constant and the compiler cannot optimize the
    // loop unrolling based on that.
    // Use a compile time constant instead.
    constexpr size_t NUM_THREADS{BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y /
                                 (THREAD_TILE_SIZE_X * THREAD_TILE_SIZE_Y)};
    size_t const thread_linear_idx{threadIdx.y * blockDim.x + threadIdx.x};

    // Cache a tile of A and B in shared memory for data reuse.
    __shared__ T
        A_thread_block_tile_transposed[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y];
    __shared__ T B_thread_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X];

    size_t const num_thread_block_tiles{(k + BLOCK_TILE_SIZE_K - 1) /
                                        BLOCK_TILE_SIZE_K};

    // Each thread in the block processes BLOCK_TILE_SIZE_Y output values.
    // Specifically, these values corresponds to
    // C[blockIdx.y * BLOCK_TILE_SIZE_Y + threadIdx.x / BLOCK_TILE_SIZE_X *
    // THREAD_TILE_SIZE_Y : blockIdx.y * BLOCK_TILE_SIZE_Y + (threadIdx.x /
    // BLOCK_TILE_SIZE_X + 1) * THREAD_TILE_SIZE_Y][blockIdx.x *
    // BLOCK_TILE_SIZE_X + threadIdx.x % BLOCK_TILE_SIZE_X *
    // THREAD_TILE_SIZE_X : blockIdx.x * BLOCK_TILE_SIZE_X + (threadIdx.x %
    // BLOCK_TILE_SIZE_X + 1) * THREAD_TILE_SIZE_X]
    T C_thread_results[THREAD_TILE_SIZE_Y][THREAD_TILE_SIZE_X] = {
        static_cast<T>(0)};
    // A_vals is cached in the register.
    T A_vals[THREAD_TILE_SIZE_Y] = {static_cast<T>(0)};
    // B_vals is cached in the register.
    T B_vals[THREAD_TILE_SIZE_X] = {static_cast<T>(0)};

    constexpr size_t NUM_VECTOR_UNITS{sizeof(int4) / sizeof(T)};
    static_assert(sizeof(int4) % sizeof(T) == 0U);
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    static_assert(BLOCK_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_X{THREAD_TILE_SIZE_X /
                                                   NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);

    for (size_t thread_block_tile_idx{0U};
         thread_block_tile_idx < num_thread_block_tiles;
         ++thread_block_tile_idx)
    {
        load_data_from_global_memory_to_shared_memory_transposed_vectorized<
            T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y, BLOCK_TILE_SIZE_K,
            NUM_THREADS>(A, lda, B, ldb, A_thread_block_tile_transposed,
                         B_thread_block_tile, thread_block_tile_idx,
                         thread_linear_idx, m, n, k);
        __syncthreads();

#pragma unroll
        for (size_t k_i{0U}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
        {
            size_t const A_thread_block_tile_row_idx{
                thread_linear_idx / (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                THREAD_TILE_SIZE_Y};
            size_t const A_thread_block_tile_col_idx{k_i};

#pragma unroll
            for (size_t thread_tile_row_idx{0U};
                 thread_tile_row_idx < THREAD_TILE_SIZE_Y;
                 ++thread_tile_row_idx)
            {
                A_vals[thread_tile_row_idx] =
                    A_thread_block_tile_transposed[A_thread_block_tile_col_idx]
                                                  [A_thread_block_tile_row_idx +
                                                   thread_tile_row_idx];
            }

            size_t const B_thread_block_tile_row_idx{k_i};
            size_t const B_thread_block_tile_col_idx{
                thread_linear_idx % (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                THREAD_TILE_SIZE_X};
// Although the read from A_thread_block_tile cannot be vectorized, the read
// from B_thread_block_tile can be vectorized.
#pragma unroll
            for (size_t thread_tile_col_vector_idx{0U};
                 thread_tile_col_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
                 ++thread_tile_col_vector_idx)
            {
                *reinterpret_cast<int4*>(
                    &B_vals[thread_tile_col_vector_idx * NUM_VECTOR_UNITS]) =
                    *reinterpret_cast<int4 const*>(
                        &B_thread_block_tile[B_thread_block_tile_row_idx]
                                            [B_thread_block_tile_col_idx +
                                             thread_tile_col_vector_idx *
                                                 NUM_VECTOR_UNITS]);
            }

            for (size_t thread_tile_row_idx{0U};
                 thread_tile_row_idx < THREAD_TILE_SIZE_Y;
                 ++thread_tile_row_idx)
            {
                for (size_t thread_tile_col_idx{0U};
                     thread_tile_col_idx < THREAD_TILE_SIZE_X;
                     ++thread_tile_col_idx)
                {
                    C_thread_results[thread_tile_row_idx]
                                    [thread_tile_col_idx] +=
                        A_vals[thread_tile_row_idx] *
                        B_vals[thread_tile_col_idx];
                }
            }
        }
        __syncthreads();
    }

    // Vectorized writing the results to DRAM.
    for (size_t thread_tile_row_idx{0U};
         thread_tile_row_idx < THREAD_TILE_SIZE_Y; ++thread_tile_row_idx)
    {
        for (size_t thread_tile_col_vector_idx{0U};
             thread_tile_col_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
             ++thread_tile_col_vector_idx)
        {
            size_t const C_row_idx{
                blockIdx.y * BLOCK_TILE_SIZE_Y +
                thread_linear_idx / (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                    THREAD_TILE_SIZE_Y +
                thread_tile_row_idx};
            size_t const C_col_idx{
                blockIdx.x * BLOCK_TILE_SIZE_X +
                thread_linear_idx % (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                    THREAD_TILE_SIZE_X +
                thread_tile_col_vector_idx * NUM_VECTOR_UNITS};
            // Vectorized read from C.
            int4 C_row_vector_vals{*reinterpret_cast<int4 const*>(
                &C[C_row_idx * ldc + C_col_idx])};
            // Vectorized read from C_thread_results.
            int4 const C_thread_results_row_vector_vals{
                *reinterpret_cast<int4 const*>(
                    &C_thread_results[thread_tile_row_idx]
                                     [thread_tile_col_vector_idx *
                                      NUM_VECTOR_UNITS])};
            // Update the values in C_row_vector_vals
            for (size_t i{0U}; i < NUM_VECTOR_UNITS; ++i)
            {
                reinterpret_cast<T*>(&C_row_vector_vals)[i] =
                    alpha * reinterpret_cast<T const*>(
                                &C_thread_results_row_vector_vals)[i] +
                    beta * reinterpret_cast<T const*>(&C_row_vector_vals)[i];
            }
            // Vectorized write to C.
            if (C_row_idx < m && C_col_idx < n)
            {
                // No need to mask out the out-of-bound invalid elements,
                // because the row of C matrix is 32-byte aligned.
                *reinterpret_cast<int4*>(&C[C_row_idx * ldc + C_col_idx]) =
                    C_row_vector_vals;
            }
        }
    }
}

template <typename T>
void launch_gemm_kernel_v05_vectorized(size_t m, size_t n, size_t k,
                                       T const* alpha, T const* A, size_t lda,
                                       T const* B, size_t ldb, T const* beta,
                                       T* C, size_t ldc, hipStream_t stream)
{
    // Feel free to play with the block tile sizes.
    // The algorithm correctness should always be guaranteed.
    constexpr unsigned int BLOCK_TILE_SIZE_X{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_Y{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_K{16U};
    // Each thread computes THREAD_TILE_SIZE_X * THREAD_TILE_SIZE_Y values of C.
    constexpr unsigned int THREAD_TILE_SIZE_X{8U};
    constexpr unsigned int THREAD_TILE_SIZE_Y{8U};
    constexpr unsigned int NUM_THREADS_PER_BLOCK{
        BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y /
        (THREAD_TILE_SIZE_X * THREAD_TILE_SIZE_Y)};
    static_assert(BLOCK_TILE_SIZE_X % THREAD_TILE_SIZE_X == 0U);
    static_assert(BLOCK_TILE_SIZE_Y % THREAD_TILE_SIZE_Y == 0U);
    static_assert(NUM_THREADS_PER_BLOCK % BLOCK_TILE_SIZE_K == 0U);
    static_assert(NUM_THREADS_PER_BLOCK % BLOCK_TILE_SIZE_X == 0U);
    static_assert(
        BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K % NUM_THREADS_PER_BLOCK == 0U);
    static_assert(
        BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y % NUM_THREADS_PER_BLOCK == 0U);
    dim3 const block_dim{NUM_THREADS_PER_BLOCK, 1U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + BLOCK_TILE_SIZE_X - 1U) /
            BLOCK_TILE_SIZE_X,
        (static_cast<unsigned int>(m) + BLOCK_TILE_SIZE_Y - 1U) /
            BLOCK_TILE_SIZE_Y,
        1U};
    gemm_v05_vectorized<T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y,
                        BLOCK_TILE_SIZE_K, THREAD_TILE_SIZE_X,
                        THREAD_TILE_SIZE_Y>
        <<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B, ldb,
                                              *beta, C, ldc);
    CHECK_LAST_CUDA_ERROR();
}

// Explicit instantiation.
template void launch_gemm_kernel_v05_vectorized<float>(
    size_t m, size_t n, size_t k, float const* alpha, float const* A,
    size_t lda, float const* B, size_t ldb, float const* beta, float* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v05_vectorized<double>(
    size_t m, size_t n, size_t k, double const* alpha, double const* A,
    size_t lda, double const* B, size_t ldb, double const* beta, double* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v05_vectorized<__half>(
    size_t m, size_t n, size_t k, __half const* alpha, __half const* A,
    size_t lda, __half const* B, size_t ldb, __half const* beta, __half* C,
    size_t ldc, hipStream_t stream);